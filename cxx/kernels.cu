#include "hip/hip_runtime.h"
//
// Created by duncan on 09-02-23.
//

#include "kernels.cuh"

#define uchar unsigned char

struct dims {
    long b;
    long c;
    long w;
    long h;
};

/**
 * This kernel performs a chained crop-interpolate operation. This means the batch of input images will be cropped to
 * a certain size, and then interpolated with a closest pixel algorithm.
 * The input image array is assumed to be of format [B, C, X, Y] in unsigned characters.
 * The output image array will be [B, C, x, y] in floats.
 * The output image has scaled the input values from [0, 255] to [0, 1]
 * It supports any amount of channels, but the output x, y is limited to the maximum thread-block size on the device.
 */
__global__ void ci_perpixel_kernel(uchar *images, struct dims in_dims, float *transforms,
                                   float *output_images, struct dims out_dims) {

    size_t batch_id = blockIdx.x;
    size_t x = threadIdx.x;
    size_t y = threadIdx.y;

    float *transform = &transforms[batch_id * 4];
    size_t image_size = in_dims.w * in_dims.h * in_dims.c;
    uchar *image = &images[batch_id * image_size];

    size_t output_image_size = out_dims.w * out_dims.h * out_dims.c;
    float *output_image = &output_images[batch_id * output_image_size];

    auto avx = float(in_dims.w - out_dims.w);  // Available space X
    auto avy = float(in_dims.h - out_dims.h);  // Available space Y

    // Get the selected box starting point as subset of the image.
    // We now scale the   0,1   range to    0,(size-64)  so we cannot have invalid percentages.
    auto x1 = transform[0] * avx;
    auto y1 = transform[1] * avy;

    // We scale the image to be 64 for zoom=0, and image_size for zoom=1
    // Clip max size based on available remaining pixels
    auto x_size = min(in_dims.w - x1, (transform[2] * avx) + out_dims.w);
    auto y_size = min(in_dims.h - y1, (transform[3] * avy) + out_dims.h);

    int sx = int(x1 + (x_size / out_dims.w) * x);
    int sy = int(y1 + (y_size / out_dims.h) * y);

    for (long c = 0; c < out_dims.c; c++) {
        output_image[(c * out_dims.w + x) * out_dims.h + y] = float(image[(c * in_dims.w + sx) * in_dims.h + sy]) / 255;
    }
}

void initialize() {
    hipFree(0);
}

torch::Tensor call_ci_kernel(const torch::Tensor &images,
                             const torch::Tensor &transforms,
                             const torch::Tensor &dims) {
    // There are no checks in place for CUDA-hosted tensors for now.
    // This can easily be implemented later.
    assert(images.device().is_cuda());
    assert(transforms.device().is_cuda());

    // Set current CUDA device to the one the tensors are residing
    hipSetDevice(images.device().index());

    /*
     * Store in and output dimensions
     */
    struct dims input_dims = {
            images.size(0),
            images.size(1),
            images.size(2),
            images.size(3)
    };
    struct dims output_dims = {
            images.size(0),
            images.size(1),
            dims[0].item().toLong(),
            dims[1].item().toLong()
    };

    /*
     * Initialize output tensor
     */
    std::vector<long> vec = {
            long(output_dims.b), long(output_dims.c),
            long(output_dims.w), long(output_dims.h)
    };
    torch::IntArrayRef out_dims_torch(vec);
    auto options = torch::TensorOptions().dtype(torch::kFloat).device(images.device());
    auto output_images = torch::zeros(out_dims_torch, options);


    // Wait for memory to finish copying
    hipDeviceSynchronize();

    /*
     * Process kernel
     */
    dim3 blocks = {uint(output_dims.w), uint(output_dims.h)};
    dim3 grids = {uint(input_dims.b)};
    ci_perpixel_kernel<<<grids, blocks>>>(
            images.data_ptr<uchar>(),
            input_dims,
            transforms.data_ptr<float>(),
            output_images.data_ptr<float>(),
            output_dims
    );

    hipDeviceSynchronize();

    return output_images;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("crop_interpolate", &call_ci_kernel, "Crop Interpolate");
}
